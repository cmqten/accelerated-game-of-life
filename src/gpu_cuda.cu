#include "hip/hip_runtime.h"
#include <game_of_life.hpp>
#include <util.hpp>

__global__ void gpu_cuda(char* world, char* buf, int width, int height)
{
    int start_y = blockIdx.x * blockDim.y + threadIdx.y;
    int stride = gridDim.x * blockDim.y;
    int x = threadIdx.x;

    for (int y = start_y; y < height; y += stride) {
        int idx = y * width + x;
        int current_cell = world[idx];

        int x_west = x ? x - 1 : width - 1;
        int x_east = x == width - 1 ? 0 : x + 1;
        int y_north = y ? y - 1 : height - 1;
        int y_south = y == height - 1 ? 0 : y + 1;

        char* north_row_ptr = world + (y_north * width);
        char* current_row_ptr = world + (y * width);
        char* south_row_ptr = world + (y_south * width);

        int neighbors = north_row_ptr[x_east] + north_row_ptr[x] + north_row_ptr[x_west] +
                        current_row_ptr[x_east] + current_row_ptr[x_west] +
                        south_row_ptr[x_east] + south_row_ptr[x] + south_row_ptr[x_west];
        int new_cell = ((neighbors == 3) || (current_cell && neighbors == 2)) & 1;
        buf[idx] = new_cell;
    }
}

double run_game_of_life_gpu(char* world, int width, int height, int gens)
{
    int size = width * height;
    char* world_d;
    char* buf_d;
    hipMalloc((void**)&world_d, size);
    hipMalloc((void**)&buf_d, size);
    hipMemcpy(world_d, world, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    int max_threads_per_block = 1024;
    int block_width = width;
    int block_height = 0;

    if (width > max_threads_per_block / 2) {
        block_height = 1;
    }
    else {
        block_height = max_threads_per_block / width;
    }

    dim3 dimBlock(block_width, block_height);

    my_timer timer;
    timer.start();
    for (int i = 0; i < gens / 2; i++) {
        gpu_cuda<<<36, dimBlock>>>(world_d, buf_d, width, height);
        gpu_cuda<<<36, dimBlock>>>(buf_d, world_d, width, height);
    }
    if (gens % 2) {
        gpu_cuda<<<36, dimBlock>>>(world_d, buf_d, width, height);
    }
    hipDeviceSynchronize();

    if (gens % 2) {
        hipMemcpy(world, buf_d, size, hipMemcpyDeviceToHost);
    }
    else {
        hipMemcpy(world, world_d, size, hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    return timer.stop();
}
