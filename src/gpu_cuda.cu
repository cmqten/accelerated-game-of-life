#include "hip/hip_runtime.h"
#include <game_of_life.hpp>
#include <util.hpp>

__global__ void gpu_cuda(char* world, char* buf, int width, int height)
{
    int start_y = blockIdx.x * blockDim.y + threadIdx.y;
    int stride = gridDim.x * blockDim.y;
    int x = threadIdx.x;

    for (int y = start_y; y < height; y += stride) {
        int idx = y * width + x;
        int current_cell = world[idx];

        int x_west = x ? x - 1 : width - 1;
        int x_east = x == width - 1 ? 0 : x + 1;
        int y_north = y ? y - 1 : height - 1;
        int y_south = y == height - 1 ? 0 : y + 1;

        char* north_row_ptr = world + (y_north * width);
        char* current_row_ptr = world + (y * width);
        char* south_row_ptr = world + (y_south * width);

        int neighbors = north_row_ptr[x_east] + north_row_ptr[x] + north_row_ptr[x_west] +
                        current_row_ptr[x_east] + current_row_ptr[x_west] +
                        south_row_ptr[x_east] + south_row_ptr[x] + south_row_ptr[x_west];
        int new_cell = ((neighbors == 3) || (current_cell && neighbors == 2)) & 1;
        buf[idx] = new_cell;
    }
}

void gpu_cuda_hip(char* world, int width, int height, int gens, double* compute_time, double* transfer_in_time, 
    double* transfer_out_time)
{
    int size = width * height;
    char* world_d;
    char* buf_d;
    my_timer timer;
    double compute_time_local;
    double transfer_in_time_local;
    double transfer_out_time_local;

    hipMalloc((void**)&world_d, size);
    hipMalloc((void**)&buf_d, size);
    timer.start();
    hipMemcpy(world_d, world, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    transfer_in_time_local = timer.stop();
    if (transfer_in_time) {
        *transfer_in_time = transfer_in_time_local;
    }

    int max_threads_per_block = 1024;
    int block_width = width;
    int block_height = 0;

    if (width > max_threads_per_block / 2) {
        block_height = 1;
    }
    else {
        block_height = max_threads_per_block / width;
    }

    dim3 dimBlock(block_width, block_height);

    timer.start();
    for (int i = 0; i < gens / 2; i++) {
        gpu_cuda<<<36, dimBlock>>>(world_d, buf_d, width, height);
        gpu_cuda<<<36, dimBlock>>>(buf_d, world_d, width, height);
    }
    if (gens % 2) {
        gpu_cuda<<<36, dimBlock>>>(world_d, buf_d, width, height);
    }
    hipDeviceSynchronize();
    compute_time_local = timer.stop();
    if (compute_time) {
        *compute_time = compute_time_local;
    }

    timer.start();
    if (gens % 2) {
        hipMemcpy(world, buf_d, size, hipMemcpyDeviceToHost);
    }
    else {
        hipMemcpy(world, world_d, size, hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    transfer_out_time_local = timer.stop();
    if (transfer_out_time) {
        *transfer_out_time = transfer_out_time_local;
    }
}
